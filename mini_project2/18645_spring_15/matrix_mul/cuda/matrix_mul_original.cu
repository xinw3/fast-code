#include "hip/hip_runtime.h"
/*
    Copyright (C) 2011  Abhinav Jauhri (abhinav.jauhri@gmail.com), Carnegie Mellon University - Silicon Valley 

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "matrix_mul.h"
#define TILE_WIDTH 2

namespace cuda
{
  __global__ 
  void 
  matrix_mul_kernel(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, int sq_dimension)
  {
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    float sum = 0.0f;
    
    for(int k = 0; k < sq_dimension; k++)
      {
	sum += sq_matrix_1[ty*sq_dimension + k] * sq_matrix_2[k*sq_dimension + tx];
      }
    sq_matrix_result[ty*sq_dimension + tx] = sum;
    
  }
  
  void 
  matrix_multiplication(float *sq_matrix_1, float *sq_matrix_2, float *sq_matrix_result, unsigned int sq_dimension)
  {
    int size = sq_dimension * sq_dimension * sizeof(float);
    float *sq_matrix_1_d, *sq_matrix_2_d, *sq_matrix_result_d;
    
    /***************************************************
  1st Part: Allocation of memory on device memory  
    ****************************************************/
    
    /* copy sq_matrix_1 and sq_matrix_2 to device memory */
    hipMalloc((void**) &sq_matrix_1_d, size);
    hipMemcpy(sq_matrix_1_d, sq_matrix_1, size, hipMemcpyHostToDevice);
    hipMalloc((void**) &sq_matrix_2_d, size);
    hipMemcpy(sq_matrix_2_d, sq_matrix_2, size, hipMemcpyHostToDevice);
    
    /*allocate sq_matrix_result on host */
    hipMalloc((void**) &sq_matrix_result_d, size);
    
    /***************************************************
   2nd Part: Inovke kernel 
    ****************************************************/
    dim3 dimBlock(sq_dimension, sq_dimension);
    dim3 dimGrid(1,1);
    matrix_mul_kernel<<<dimGrid, dimBlock, dimBlock.x * dimBlock.x * sizeof(float)>>>(sq_matrix_1_d, sq_matrix_2_d, sq_matrix_result_d, sq_dimension);
    
    /***************************************************
   3rd Part: Transfer result from device to host 
    ****************************************************/
    hipMemcpy(sq_matrix_result, sq_matrix_result_d, size, hipMemcpyDeviceToHost);
    hipFree(sq_matrix_1_d);
    hipFree(sq_matrix_2_d);
    hipFree(sq_matrix_result_d);
  }  
} // namespace cuda
